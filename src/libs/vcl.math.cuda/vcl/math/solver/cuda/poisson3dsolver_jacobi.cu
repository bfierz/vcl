#include "hip/hip_runtime.h"
/*
 * This file is part of the Visual Computing Library (VCL) release under the
 * MIT license.
 *
 * Copyright (c) 2018 Basil Fierz
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy
 * of this software and associated documentation files (the "Software"), to deal
 * in the Software without restriction, including without limitation the rights
 * to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 * copies of the Software, and to permit persons to whom the Software is
 * furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in all
 * copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 * SOFTWARE.
 */
#include <vcl/core/cuda/common.inc>
#include <vcl/core/cuda/math.inc>

extern "C"
__global__ void PoissonUpdateSolution
(
	const unsigned int X,
	const unsigned int Y,
	const unsigned int Z,

	const float* __restrict__ Ac,
	const float* __restrict__ Ax_l,
	const float* __restrict__ Ax_r,
	const float* __restrict__ Ay_l,
	const float* __restrict__ Ay_r,
	const float* __restrict__ Az_l,
	const float* __restrict__ Az_r,
	const float* __restrict__ rhs,

	float* __restrict__ unknowns,
	float* __restrict__ next,
	float* __restrict__ error
)
{
	// x^{n+1} = D^-1 (b - R x^{n})
	//                -------------
	//                      q

	const unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	const unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;
	const unsigned int z = blockIdx.z*blockDim.z + threadIdx.z;
	const unsigned int index = X*Y * z + X * y + x;

	if (0 < x && x < X-1 &&
		0 < y && y < Y-1 &&
		0 < z && z < Z-1   )
	{
		float q =
			unknowns[index - 1] * Ax_l[index] +
			unknowns[index + 1] * Ax_r[index] +
			unknowns[index - X] * Ay_l[index] +
			unknowns[index + X] * Ay_r[index] +
			unknowns[index - X * Y] * Az_l[index] +
			unknowns[index + X * Y] * Az_r[index];

		const float c = Ac[index];
		float n = (rhs[index] - q) / c;
		n = (c != 0) ? n : unknowns[index];

		next[index] = n;

		// Compute the error
		if (c)
		{
			const float e = rhs[index] - (Ac[index] * unknowns[index] + q);
			atomicAdd(error, e * e);
		}
	}
}
